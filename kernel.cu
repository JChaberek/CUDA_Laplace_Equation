#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include <cstdlib>
#include <stdlib.h>
#include <conio.h>
#include <ctype.h>
#include <hip/device_functions.h>
#include <string.h>
#include <inttypes.h>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <Python.h>

#define potential 1


//print area in the console
void print(int size,double *tab) {	
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			printf(" %.3f ", tab[y * size + x]);
		}
		printf("\n");
	}
}


//set electric potential in area to zero
__global__ void Initialization_kernel(int size,double *area_gpu) { 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	
	area_gpu[x*size + y] = 0;
}


//initialize Core in area
__global__ void Core_kernel(int size, int center,double* area_gpu, int* radius) { 
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	double x_dis = abs((((x + size) % size)-center));
	double y_dis = abs((y - center));
	double r = *radius;
	
	//set core potential
	if ((x_dis*x_dis)+(y_dis*y_dis) <= r*r) {
		area_gpu[x*size + y] = potential;		
	}
}


//calculate potential in the sub-area
__global__ void Calculation_kernel(double *epsilon, int center, int size, int *r, int *R , double *Buffor, double *area ) {
	int diff = *R - *r;
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int Rr = (*R) * (*R);
	double x_dis = abs((((x + size) % size) - center));
	double y_dis = abs((y - center));
	Buffor[x*size + y] = area[x*size + y];
	
	if ((x_dis*x_dis+y_dis*y_dis <= Rr)) {
		__syncthreads();
		
		for (int layer = 1; layer <= diff; layer++) {
			Buffor[x * size + y] = area[x * size + y];
			__syncthreads();
			double r_small = *r + layer - 1;
			double R_big = *r + layer;
			
			if (((x_dis*x_dis)+(y_dis*y_dis) > (r_small*r_small)) && (((x_dis*x_dis)+(y_dis*y_dis)) <= (R_big*R_big))) {
				
				//Numeric Laplace Equation
				area[x * size + y] = (Buffor[x * size + y - 1] + Buffor[x * size + y + 1] + Buffor[x * (size)+y - size] + Buffor[x * (size)+y + size]) / 4;
				__syncthreads();

				//calculate difference between actual and previous iteration
				if (area[x * size + y] >= 0.000001) {
					*epsilon = abs(area[x * size + y] - Buffor[x * size + y]);
				}
				__syncthreads();
			}
		}
		__syncthreads();
	}
}


//save results to csv file
void create_csv(int size, char* filename, double* area, int area_radius) {
	char* filename1;
	char str1[] = "Radius";
	FILE *fp, *fp1;
	filename = strcat(filename, ".csv");
	fp = fopen(filename, "w+");
	filename1 = strcat(str1, ".csv");
	fp1 = fopen(filename1, "w+");
	
	if (fp == NULL){
		printf("Unable to create a file.\n");
		exit(EXIT_FAILURE);
	}
	
	if (fp1 == NULL){
		printf("Unable to create a file.\n");
		exit(EXIT_FAILURE);
	}
	
	int newline = 0;
	for (int i = 0; i < size*size; i++) {
		
		if (((i + size) % size == 0) && (newline > 0)) {
			fprintf(fp, "\n");
			fprintf(fp, "%f,", area[i]);
		}
		
		else{ 
		fprintf(fp, "%f,", area[i]); 
		}
		newline++;
	}
	
	fprintf(fp1, "%i", area_radius);
	fclose(fp1);
	fclose(fp);
	printf("\n %s, %s files created", filename, filename1);
}


int main(){
	const int size = 32;
	const int n = 1024;
	const int center = n/2;
	bool exit_program = false;
	int choice = 10;
	int radius;
	int area_radius = 0;
	int* R_area_gpu, *r_gpu;
	double *area_gpu, *area_cpu, *Buffor ,*Buffor_cpu, *epsilon_g;
	char str[] = "Data";
	double epsilon_c = 1;
	unsigned long long int iteration = 0;
	dim3 dimblock(size, size);
	dim3 dimGrid(n / size, n / size);
	area_cpu = (double*)malloc(n * n * sizeof(double));
	Buffor_cpu = (double*)malloc(n * n * sizeof(double));
	hipMalloc((void**)&area_gpu, n * n * sizeof(double));
	hipMalloc((void**)&epsilon_g, sizeof(double));
	hipMalloc((void**)&R_area_gpu, sizeof(int));
	hipMalloc((void**)&Buffor, n * n * sizeof(double));
	hipMalloc((void**)&r_gpu, sizeof(int));
	
	//program menu
	while (exit_program == false) {
		system("cls");
		printf("Choose operation\n");
		printf("Quit program - 0\n");
		printf("Initialize area - 1\n");
		printf("Calculate the core - 2\n");
		printf("Calculate area - 3\n");
		printf("Print actual area - 5\n");
		printf("Save into CSV file - 6\n");
		printf("Visualisation - 7\n");
		scanf_s("%i", &choice);

		if (choice == 0) {
			hipFree(area_gpu);
			free(area_cpu);
			hipFree(r_gpu);
			hipFree(R_area_gpu);
			hipFree(Buffor);
			free(Buffor_cpu);
			hipFree(epsilon_g);
			exit_program = true;
		}
	

		if (choice == 1) {
			hipMemcpy(Buffor, Buffor_cpu, n * n * sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(area_gpu, area_cpu, n * n * sizeof(double), hipMemcpyHostToDevice);
			Initialization_kernel << <dimblock, dimGrid >> > (n,area_gpu);
			Initialization_kernel << <dimblock, dimGrid >> > (n,Buffor);
			hipMemcpy(area_cpu, area_gpu, n * n * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(Buffor_cpu, Buffor, n * n * sizeof(double), hipMemcpyDeviceToHost);
		}


		if (choice == 3) {
			iteration = 0;
			epsilon_c = 1;
			printf("\nArea radius: ");
			scanf_s("%i", &area_radius);
			hipMemcpy(area_gpu, area_cpu,n * n * sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(R_area_gpu, &area_radius, sizeof(int), hipMemcpyHostToDevice);
			while (epsilon_c > 0.000000015) {
				hipMemcpy(epsilon_g, &epsilon_c, sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(Buffor, Buffor_cpu, n * n * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(area_gpu, area_cpu, n * n * sizeof(double), hipMemcpyHostToDevice);
				Calculation_kernel << < dimblock, dimGrid >> > (epsilon_g, center, n, r_gpu, R_area_gpu, Buffor, area_gpu);
				hipMemcpy(Buffor_cpu, Buffor, n * n * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(area_cpu, area_gpu, n * n * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(&epsilon_c, epsilon_g, sizeof(double), hipMemcpyDeviceToHost);
				iteration = iteration + 1;
				if (iteration > n*2) {
					epsilon_c = 0;
				}
			}
			
			printf("iterations %llu\n", iteration);
			printf("calculations done..\n");
			char mychar;
			scanf("%c", &mychar);
			getchar();
			iteration = 0;
			epsilon_c = 1;
		}


		if (choice == 5) {
			printf("Electrical potential for this area \n");
			print(n, area_cpu);
			printf("calculations done..\n");
			char mychar;
			scanf("%c", &mychar);
			getchar();
		}


		if (choice == 2) {
			printf("Core radius: ");
			scanf_s("%i", &radius);
			hipMemcpy(r_gpu, &radius, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(area_gpu, area_cpu, n * n * sizeof(double), hipMemcpyHostToDevice);
			Core_kernel << <dimblock,dimGrid >> > (n,center,area_gpu, r_gpu);
			hipMemcpy(area_cpu, area_gpu, n * n * sizeof(double), hipMemcpyDeviceToHost);
		}


		if (choice == 6) {
			char str[] = "Data";
			create_csv(n, str, area_cpu, area_radius);
			char mychar;
			scanf("%c", &mychar);
			getchar();
		}

		//Python visualisation
		if (choice == 7) {
			char path[] = "simulation.py";
			FILE* fp;
			int argc = 1;
			wchar_t* argv[1];
			argv[0] = L"simulation.py";
			Py_Initialize();
			Py_SetProgramName(argv[0]);
			PySys_SetArgv(argc, argv);
			fp = _Py_fopen(path, "r");
			PyRun_SimpleFile(fp, path);
			Py_Finalize();
			_getch();
		}
	}
    
	return 0;
}


